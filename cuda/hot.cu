#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hot.h"
#include "../hot_interface.h"
#include "../../cuda/shared.h"
#include "../../profiler.h"
#include "../../comms.h"
#include "kernels.k"

// Performs the CG solve, you always want to perform these steps, regardless
// of the context of the problem etc.
void solve_diffusion_2d(
    const int nx, const int ny, Mesh* mesh, const double dt, double* x, 
    double* r, double* p, double* rho, double* s_x, double* s_y, 
    double* Ap, int* end_niters, double* end_error, double* reduce_array,
    const double* edgedx, const double* edgedy)
{
  // Store initial residual
  double local_old_r2 = initialise_cg(
      nx, ny, dt, p, r, x, rho, s_x, s_y, reduce_array, edgedx, edgedy);

  double global_old_r2 = reduce_all_sum(
      local_old_r2);

  handle_boundary_2d(nx, ny, mesh, p, NO_INVERT, PACK);
  handle_boundary_2d(nx, ny, mesh, x, NO_INVERT, PACK);

  // TODO: Can one of the allreduces be removed with kernel fusion?
  int ii = 0;
  for(ii = 0; ii < MAX_INNER_ITERATIONS; ++ii) {

    const double local_pAp = calculate_pAp(nx, ny, s_x, s_y, p, Ap, reduce_array);
    const double global_pAp = reduce_all_sum(local_pAp);
    const double alpha = global_old_r2/global_pAp;

    const double local_new_r2 = calculate_new_r2(nx, ny, alpha, x, p, r, Ap, reduce_array);
    const double global_new_r2 = reduce_all_sum(local_new_r2);
    const double beta = global_new_r2/global_old_r2;
    handle_boundary_2d(nx, ny, mesh, x, NO_INVERT, PACK);

#if 0
    // Check if the solution has converged
    if(fabs(global_new_r2) < 1.0e-10) {
      global_old_r2 = global_new_r2;
      break;
    }
#endif // if 0

    update_conjugate(nx, ny, beta, r, p);
    handle_boundary_2d(nx, ny, mesh, p, NO_INVERT, PACK);

    // Store the old squared residual
    global_old_r2 = global_new_r2;
  }

  *end_niters = ii;
  *end_error = global_old_r2;
}

// Initialises the CG solver
double initialise_cg(
    const int nx, const int ny, const double dt, double* p, double* r,
    const double* x, const double* rho, double* s_x, double* s_y, double* reduce_array,
    const double* edgedx, const double* edgedy)
{
  int nblocks = ceil((nx+1)*ny/(double)NTHREADS);
  calc_s_x<<<nblocks, NTHREADS>>>(
      nx, ny, dt, s_x, rho, edgedx);
  gpu_check(hipDeviceSynchronize());

  nblocks = ceil(nx*(ny+1)/(double)NTHREADS);
  calc_s_y<<<nblocks, NTHREADS>>>(
      nx, ny, dt, s_y, rho, edgedy);
  gpu_check(hipDeviceSynchronize());

  nblocks = ceil(nx*ny/(double)NTHREADS);
  calc_initial_r2<<<nblocks, NTHREADS>>>(
      nx, ny, s_x, s_y, x, p, r, reduce_array);
  gpu_check(hipDeviceSynchronize());

  double initial_r2 = 0.0;
  finish_sum_reduce(nblocks, reduce_array, &initial_r2);
  return initial_r2;
}

// Calculates a value for alpha
double calculate_pAp(
    const int nx, const int ny, const double* s_x, 
    const double* s_y, double* p, double* Ap, double* reduce_array)
{
  START_PROFILING(&compute_profile);
  int nblocks = ceil(nx*ny/(double)NTHREADS);
  calc_pAp<<<nblocks, NTHREADS>>>(
      nx, ny, s_x, s_y, p, Ap, reduce_array);
  gpu_check(hipDeviceSynchronize());

  double pAp = 0.0;
  finish_sum_reduce(nblocks, reduce_array, &pAp);
  STOP_PROFILING(&compute_profile, "calculate alpha");
  return pAp;
}

// Updates the current guess using the calculated alpha
double calculate_new_r2(
    int nx, int ny, double alpha, double* x, double* p, double* r, 
    double* Ap, double* reduce_array)
{
  START_PROFILING(&compute_profile);

  int nblocks = ceil(nx*ny/(double)NTHREADS);
  calc_new_r2<<<nblocks, NTHREADS>>>(nx, ny, alpha, x, p, r, Ap, reduce_array);
  gpu_check(hipDeviceSynchronize());

  double new_r2 = 0.0;
  finish_sum_reduce(nblocks, reduce_array, &new_r2);
  STOP_PROFILING(&compute_profile, "calculate new r2");
  return new_r2;
}

// Updates the conjugate from the calculated beta and residual
void update_conjugate(
    const int nx, const int ny, const double beta, const double* r, double* p)
{
  START_PROFILING(&compute_profile);

  int nblocks = ceil(nx*ny/(double)NTHREADS);
  update_p<<<nblocks, NTHREADS>>>(nx, ny, beta, r, p);
  gpu_check(hipDeviceSynchronize());

  STOP_PROFILING(&compute_profile, "update conjugate");
}

// Prints the vector to std out
void print_vec(
    const int nx, const int ny, double* a)
{
  for(int ii = 0; ii < ny; ++ii) {
    for(int jj = 0; jj < nx; ++jj) {
      printf("%.3e ", a[ii*nx+jj]);
    }
    printf("\n");
  }
}

